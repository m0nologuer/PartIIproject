#include "hip/hip_runtime.h"
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math_functions.h>
#include <>
#include "ParticleContainer.h"
#ifdef USE_CUDA  

extern "C" __device__ float W_poly6_cuda(Particle::vec3 r, float h)
{
	double radius = r.x*r.x + r.y*r.y + r.z*r.z;

	if (radius < h_squared)
	{
		//constant is 315/64pi
		double result = pow(h_squared - radius, 3.0) * 1.56668147 / h_9;
		return result;
	}
	else //ignore particles outside a certain large radius
		return 0;
}
extern "C" __device__ Particle::vec3 dW_poly6_cuda(Particle::vec3 r, float h)
{
	double radius_2 = r.x*r.x + r.y*r.y + r.z*r.z;

	if (radius_2 < h_squared)
	{
		//constant is 315/64pi
		double radius = sqrt(radius_2);
		double result = -6 * radius * pow(h_squared - radius_2, 2.0) * 1.56668147 / h_9;
		Particle::vec3 grad;
		grad.x = r.x * result;
		grad.y = r.y * result;
		grad.z = r.z * result;
		return grad;
	}
	else //ignore particles outside a certain large radius
	{
		Particle::vec3 zero;
		zero.x = 0;
		zero.y = 0;
		zero.z = 0;
		return zero;
	}
}
extern "C" __device__ Particle::vec3 dW_spiky_cuda(Particle::vec3 r, float h)
{
	double radius_2 = r.x*r.x + r.y*r.y + r.z*r.z;

	if (radius_2 < h_squared)
	{
		//constant is 15/pi
		double radius = sqrt(radius_2);
		double result = -3 * pow(h - sqrt(radius), 2.0) * 4.77464829 / h_6;
		Particle::vec3 grad;
		grad.x = r.x * result;
		grad.y = r.y * result;
		grad.z = r.z * result;
		return grad;
	}
	else //ignore particles outside a certain large radius
	{
		Particle::vec3 zero;
		zero.x = 0;
		zero.y = 0;
		zero.z = 0;
		return zero;
	}
}
extern "C" __global__ void solverIterationPositions(Particle *particles, const int *neighbour_indexes,
	float h, float Wq, float corr_k, int n)
{
	__shared__ Particle::vec3 predicted_pos_contributions[THREADS_PER_BLOCK];

	//sort out indexes
	int id = blockDim.x * blockIdx.x + threadIdx.x; //thread id, from 0 to MAX_PARTICLE_COUNT * MAX_NEIGHBOURS
	int particle_index = id / MAX_NEIGHBOURS; //the particle we sum at
	if (particle_index > MAX_PARTICLE_COUNT)
		return;
	int storage_index = threadIdx.x;
	bool reducer = threadIdx.x % MAX_NEIGHBOURS == 0;

	//calculate contributions to positions
	Particle pi = particles[particle_index];
	if (!(pi.life > 0))
		return;

	int particle_neighbour_index = neighbour_indexes[id];
	if (particle_neighbour_index < 0) //if there's no neighbour at this location
	{
		predicted_pos_contributions[storage_index].x = 0;
		predicted_pos_contributions[storage_index].y = 0;
		predicted_pos_contributions[storage_index].z = 0;
	}
	else
	{
		Particle pj = particles[particle_neighbour_index]; //indexing by rows and columns
		
		Particle::vec3 distance;
		distance.x = (pi.predicted_pos.x - pj.predicted_pos.x);
		distance.y = (pi.predicted_pos.y - pj.predicted_pos.y);
		distance.z = (pi.predicted_pos.z - pj.predicted_pos.z);

		double s_corr = W_poly6_cuda(distance, h) / Wq;
		s_corr = -corr_k * pow(s_corr, n);

		predicted_pos_contributions[storage_index] = dW_spiky_cuda(distance, h);
		predicted_pos_contributions[storage_index].x *= (pi.lambda + pj.lambda + s_corr);
		predicted_pos_contributions[storage_index].y *= (pi.lambda + pj.lambda + s_corr);
		predicted_pos_contributions[storage_index].z *= (pi.lambda + pj.lambda + s_corr);
 	}
	__syncthreads();

	if (reducer)
	{
		float x, y,z = 0;
		for (int j = 0; j < MAX_NEIGHBOURS; j++)
		{
			x += predicted_pos_contributions[storage_index + j].x;
			y += predicted_pos_contributions[storage_index + j].y;
			z += predicted_pos_contributions[storage_index + j].z;
		}
		particles[particle_index].predicted_pos.x = x;
		particles[particle_index].predicted_pos.y = y;
		particles[particle_index].predicted_pos.z = z;

	}
}
extern "C" __global__ void solverIterationLambdas(Particle *particles, const int *neighbour_indexes,
	float p0, float h)
{
	//data shared between each block
	__shared__ float lambda_numerators[THREADS_PER_BLOCK];
	__shared__ float lambda_denominators[THREADS_PER_BLOCK];

	//sort out indexes
	int id = blockDim.x * blockIdx.x + threadIdx.x; //thread id, from 0 to MAX_PARTICLE_COUNT * MAX_NEIGHBOURS
	int particle_index = id / MAX_NEIGHBOURS; //the particle we sum at
	int neighbour_index = id % MAX_NEIGHBOURS; //the particle we sum at
	if (particle_index > MAX_PARTICLE_COUNT)
		return;
	int storage_index = threadIdx.x;
	bool reducer = threadIdx.x % MAX_NEIGHBOURS == 0;
	
	//calculate contributions to lambda
	Particle pi = particles[particle_index];
	if (!(pi.life > 0))
		return;

	int particle_neighbour_index = neighbour_indexes[id];
	if (neighbour_index < 0) //if there's no neighbour at this location
	{
		lambda_numerators[storage_index] = 0;
		lambda_denominators[storage_index] = 0;
	}
	else
	{
		Particle pj = particles[particle_neighbour_index]; //indexing by rows and columns
		Particle::vec3 distance; 
		distance.x = (pi.predicted_pos.x - pj.predicted_pos.x);
		distance.y = (pi.predicted_pos.y - pj.predicted_pos.y);
		distance.z = (pi.predicted_pos.z - pj.predicted_pos.z);

		lambda_numerators[storage_index] = W_poly6_cuda(distance, h);

		Particle::vec3 d_distance = dW_poly6_cuda(distance, h);
		lambda_denominators[storage_index] = -d_distance.x * pj.speed.x + d_distance.y * pj.speed.y
			+ d_distance.z * pj.speed.z;
	}
	__syncthreads();
	
	//reduction
	if (reducer)
	{
		float numerator, denominator = 0;
		for (int j = 0; j < MAX_NEIGHBOURS; j++)
		{
			numerator += lambda_numerators[storage_index + j];
			denominator += lambda_denominators[storage_index + j];
		}
		numerator = numerator / p0 - 1;
		denominator = denominator / p0;
		particles[particle_index].lambda = numerator / denominator;
	}
}

void ParticleContainer::intialize_CUDA()
{
	size_t mem_size = sizeof(Particle) * MAX_PARTICLE_COUNT;
	gpuErrchk(hipMalloc((void **)&container_CUDA, mem_size));

	mem_size = sizeof(int) * MAX_PARTICLE_COUNT * MAX_NEIGHBOURS;
	neighbour_array = (int*)malloc(mem_size);
	gpuErrchk(hipMalloc((void **)&neighbours_CUDA, mem_size));
} 
void ParticleContainer::cleanup_CUDA()
{
	free(neighbour_array);
	gpuErrchk(hipFree((void **)&container_CUDA));
	gpuErrchk(hipFree((void **)&neighbours_CUDA));
}
void ParticleContainer::solverIterations_CUDA(void)
{
	int start_time = glutGet(GLUT_ELAPSED_TIME);
	int time;
	//copy the data across
	size_t mem_size = sizeof(Particle) * MAX_PARTICLE_COUNT;
	gpuErrchk(hipMemcpy(container_CUDA,container, mem_size, hipMemcpyHostToDevice));
	//mem_size = sizeof(int) * MAX_PARTICLE_COUNT * MAX_NEIGHBOURS;
	//gpuErrchk(hipMemcpy(neighbours_CUDA,neighbour_array, mem_size, hipMemcpyHostToDevice));

	RECORD_SPEED("		Copy data across  %d ms \n");

	//kernel properties
	int threadsPerBlock = THREADS_PER_BLOCK; //so we can have  multiple blocks on processors
	int blocksPerGrid = MAX_PARTICLE_COUNT * MAX_NEIGHBOURS / THREADS_PER_BLOCK + 1;
	 
	for (int i = 0; i < iteration_count; i++)
	{
		// Launch the CUDA Kernel for lambdas
		solverIterationLambdas << <blocksPerGrid, threadsPerBlock >> >(container_CUDA, neighbours_CUDA, p0, h);

		//Error handling
		gpuErrchk(hipDeviceSynchronize());
		gpuErrchk(hipGetLastError());

		// Launch the CUDA Kernel for positions
		solverIterationPositions << <blocksPerGrid, threadsPerBlock >> >(container_CUDA, neighbours_CUDA, h, 
			Wq, corr_k, n);
		//Error handling
		gpuErrchk(hipDeviceSynchronize());
		gpuErrchk(hipGetLastError());
	}

	RECORD_SPEED("		Kernel iterations  %d ms \n");

	//copy the data back
	mem_size = sizeof(Particle) * MAX_PARTICLE_COUNT;
	gpuErrchk(hipMemcpy(container, container_CUDA, mem_size, hipMemcpyDeviceToHost));

	RECORD_SPEED("		Copy data back  %d ms \n");
}
#endif